#include "hip/hip_runtime.h"

#include <cstdio>

#include "cudaCalcMurmurHash64.h"
#include "cudaMurmurHash64.cuh"
#include "cudaReadUnaligned.cuh"

__global__ void kerCalcHash(
    DB::UInt32 str_num,
    char * arr,
    DB::UInt32 * begs,
    bool interpret_as_lengths,
    DB::UInt32 * lens,
    unsigned int seed,
    DB::UInt64 * res_hash)
{
    DB::UInt32 i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < str_num))
        return;

    DB::UInt32 len = lens[i], beg = begs[i];
    if (!interpret_as_lengths)
        --len;

    DB::UInt64 h = cudaMurmurHash64(&(arr[beg]), len, seed);

    /// TODO make it optional
    if (h == 0xFFFFFFFFFFFFFFFF)
        h = 0x0000000000000000;

    res_hash[i] = h;
}

void cudaCalcMurmurHash64(
    DB::UInt32 str_num,
    char * buf,
    bool interpret_as_lengths,
    DB::UInt32 * lens,
    DB::UInt32 * offsets,
    unsigned int seed,
    DB::UInt64 * res_hash,
    hipStream_t stream)
{
    kerCalcHash<<<(str_num / 256) + 1, 256, 0, stream>>>(str_num, buf, offsets, interpret_as_lengths, lens, seed, res_hash);
}