#include "hip/hip_runtime.h"

#include <cstdio>

#include <Common/Cuda/City_Hash/cudaCityHash.cuh>
#include <Common/Cuda/cudaCalcCityHash64.h>
#include <Common/Cuda/cudaReadUnaligned.cuh>

__global__ void
kerCalcHash(DB::UInt32 str_num, char * arr, DB::UInt32 * begs, bool interpret_as_lengths, DB::UInt32 * lens, DB::UInt64 * res_hash)
{
    DB::UInt32 i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < str_num))
        return;

    DB::UInt32 len = lens[i], beg = begs[i];
    if (!interpret_as_lengths)
        --len;

    DB::UInt64 h = CityHash_v1_0_2_cuda::cudaCityHash64(&(arr[beg]), len);

    /// TODO make it optional
    if (h == 0xFFFFFFFFFFFFFFFF)
        h = 0x0000000000000000;

    res_hash[i] = h;
}

void cudaCalcCityHash64(
    DB::UInt32 str_num,
    char * buf,
    bool interpret_as_lengths,
    DB::UInt32 * lens,
    DB::UInt32 * offsets,
    DB::UInt64 * res_hash,
    hipStream_t stream)
{
    kerCalcHash<<<(str_num / 256) + 1, 256, 0, stream>>>(str_num, buf, offsets, interpret_as_lengths, lens, res_hash);
}