#include "hip/hip_runtime.h"
#include <stdexcept>
#include <hipcub/hipcub.hpp>

#include <Common/Cuda/cudaReadUnaligned.cuh>
#include <Common/Cuda/cudaMurmurHash64.cuh>
#include <Common/HashTable/Cuda/CudaStringsHashTable.h>

typedef CudaStringsHashTable::SizeType  SizeType;
typedef CudaStringsHashTable::HashType  HashType;
typedef CudaStringsHashTable::Pointer   Pointer;

CudaStringsHashTable::CudaStringsHashTable(SizeType buckets_num_, SizeType str_buf_max_sz_) : 
    buckets_num(buckets_num_), str_buf_max_sz(str_buf_max_sz_), str_buf_sz(0)
{
    hashes = CudaArrayPtr<HashType>(new CudaArray<HashType>(buckets_num));
    lens = CudaArrayPtr<SizeType>(new CudaArray<SizeType>(buckets_num));
    offsets = CudaArrayPtr<SizeType>(new CudaArray<SizeType>(buckets_num));
    pointers = CudaArrayPtr<Pointer>(new CudaArray<Pointer>(buckets_num));
    str_buf = CudaArrayPtr<char>(new CudaArray<char>(str_buf_max_sz));
    buf1 = CudaArrayPtr<SizeType>(new CudaArray<SizeType>(buckets_num));
    buf2 = CudaArrayPtr<SizeType>(new CudaArray<SizeType>(buckets_num));
    buf4_sz = 0;
    hipcub::DeviceScan::ExclusiveSum(nullptr, buf4_sz, buf1->getData(), buf2->getData(), buckets_num);
    buf4 = CudaArrayPtr<char>(new CudaArray<char>(buf4_sz));
    cuda_total_added_num = CudaArrayPtr<SizeType>(new CudaArray<SizeType>(1));
    host_total_added_num = CudaHostPinnedArrayPtr<SizeType>(new CudaHostPinnedArray<SizeType>(1));
    host_table_is_full_flag = CudaHostPinnedArrayPtr<bool>(new CudaHostPinnedArray<bool>(1));
}

__global__ void kerErase(SizeType buckets_num, HashType *hashes, SizeType *table_lens, Pointer *ptrs)
{
    SizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < buckets_num)) return;
    hashes[i] = DBMS_CUDA_EMPTY_HASH_VAL;
    table_lens[i] = DBMS_CUDA_EMPTY_LEN_VAL;
    ptrs[i] = nullptr;
}

__global__ void kerFillZero(SizeType n, SizeType *arr)
{
    SizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < n)) return;   
    arr[i] = 0;
}

/// TODO it's just temporal!!
__device__ bool table_is_full = false;

/// TODO try to store own string (or part of it) in registers
__global__ void kerAddData(SizeType str_num, char *buf, SizeType *offsets, SizeType *lens, unsigned int seed,
                           SizeType buckets_num, HashType *hashes, SizeType *table_lens, Pointer *ptrs,
                           SizeType *res_buckets, SizeType *new_strings_lens)
{
    SizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < str_num)) return;

    SizeType len = lens[i], offset = offsets[i];
    if (len == DBMS_CUDA_EMPTY_LEN_VAL) return;

    HashType h = cudaMurmurHash64(&(buf[offset]), len, seed);

    /// empty hash value collision
    if (h == DBMS_CUDA_EMPTY_HASH_VAL) h++;

    SizeType    bucket_i = h%buckets_num;
    bool        hit = false, our_string = false;
    /// TODO some kind of total fill check
    while (!hit) {
        HashType   x = hashes[bucket_i];
        if (x == DBMS_CUDA_EMPTY_HASH_VAL) {
            HashType    old = atomicCAS((unsigned long long int*)&hashes[bucket_i], (unsigned long long int)DBMS_CUDA_EMPTY_HASH_VAL, (unsigned long long int)h);
            hit = ((old == DBMS_CUDA_EMPTY_HASH_VAL)||(old == h));
        } else {
            hit = (x == h);
        } 

        /// here hit==true only hash coincidence, not strings
        if (hit) {
            SizeType    len_in_bucket = table_lens[bucket_i];
            if (len_in_bucket == DBMS_CUDA_EMPTY_LEN_VAL) {
                SizeType    old_len_in_bucket = atomicCAS(&table_lens[bucket_i], DBMS_CUDA_EMPTY_LEN_VAL, len);
                if (old_len_in_bucket == DBMS_CUDA_EMPTY_LEN_VAL) {
                    len_in_bucket = len;
                } else {
                    len_in_bucket = old_len_in_bucket;
                }
            }
            if (len_in_bucket != len) hit = false;
        }

        /// here hit==true only hash and length coincidence, not strings        
        if (hit) {
            /// get current bucket string
            Pointer     ptr = ptrs[bucket_i];
            if (ptr == nullptr) {
                Pointer                 old_ptr;
                old_ptr = reinterpret_cast<Pointer>(atomicCAS((unsigned long long int*)&ptrs[bucket_i], 
                    reinterpret_cast<unsigned long long int>(nullptr),
                    reinterpret_cast<unsigned long long int>(&(buf[offset]))));
                if (old_ptr == nullptr) {
                    /// ISSUE should we optimize this case somehow? i.e. we will compare strings with our selves
                    ptr = (Pointer)&(buf[offset]);
                    our_string = true;
                } else {
                    ptr = old_ptr;
                }
            }

            /// compare strings
            const char      *data = &(buf[offset]);
            const char      *end = data + len;
            const char      *data_next = ptr;
            const char      *end_next = data_next + len;
            bool            is_first_read = true,
                            is_first_read_next = true;
            DB::UInt64      tmp_buf, tmp_buf_next;

            while(data != end)
            {
                DB::UInt64  v = cudaReadStringUnaligned64(is_first_read, tmp_buf, data, end),
                            v_next = cudaReadStringUnaligned64(is_first_read_next, tmp_buf_next, data_next, end_next);
                if (v != v_next) hit = false;   //ISSUE break?   
            }
        }

        /// linear search, if failed
        if (!hit) {
            bucket_i = (bucket_i+1)%buckets_num;
            /// TEST
            if (bucket_i == h%buckets_num) {
                //printf("table filled; fail;\n");
                table_is_full = true;
                return;
            }
            /// TEST END
        }
    }

    res_buckets[i] = bucket_i;

    if (our_string) new_strings_lens[bucket_i] = (((len-1)/8)+1)*8;
}

__global__ void kerCopyAddedStrings( SizeType str_num, char *buf, SizeType *offsets, SizeType *lens, SizeType *buckets, 
                                     SizeType old_str_buf_sz, SizeType *new_strings_offsets, 
                                     Pointer *hash_table_ptrs, char *hash_table_str_buf )
{
    SizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < str_num)) return;

    SizeType    len = lens[i], offset = offsets[i];
    if (len == DBMS_CUDA_EMPTY_LEN_VAL) return;
    SizeType    bucket_i = buckets[i];
    Pointer     hash_table_old_ptr = hash_table_ptrs[bucket_i],
                my_ptr = (Pointer)&(buf[offset]);
    if (hash_table_old_ptr == my_ptr) {
        SizeType    hash_table_offset = old_str_buf_sz + new_strings_offsets[bucket_i];

        const char     *data = &(buf[offset]);
        const char     *end = data + len;
        bool            is_first_read = true;
        DB::UInt64      tmp_buf;
        DB::UInt64     *data_res = (DB::UInt64 *)&(hash_table_str_buf[hash_table_offset]);

        while(data != end)
        {
            *data_res = cudaReadStringUnaligned64(is_first_read, tmp_buf, data, end);
            data_res++;
        }

        hash_table_ptrs[bucket_i] = &(hash_table_str_buf[hash_table_offset]);
    }
}

/// kind of stupid 1 thread kernal to calculate exclusive_scan total sum
__global__ void    kerCalcTotalAddedLen(SizeType buckets_num, SizeType *new_strings_offsets, SizeType *new_strings_lens,
                                        SizeType *res_total_added_len)
{
    *res_total_added_len = new_strings_offsets[buckets_num-1] + new_strings_lens[buckets_num-1];
}

__global__ void    kerCalcOffsets(SizeType buckets_num, Pointer str_buf, Pointer *ptrs, SizeType *res_offsets )
{
    SizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < buckets_num)) return;
    res_offsets[i] = (SizeType)(ptrs[i] - str_buf);
}

__global__ void    kerFillEmptyResBucket(SizeType elements_num, SizeType *res_buckets )
{
    SizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < elements_num)) return;
    res_buckets[i] = ~((SizeType)0);
} 

void    CudaStringsHashTable::erase(hipStream_t stream)
{
    kerErase<<<(buckets_num/256)+1,256,0,stream>>>(buckets_num, hashes->getData(), lens->getData(), pointers->getData());
    str_buf_sz = 0;
}

/// TODO offsets and lens interfere with class fields
void    CudaStringsHashTable::addData(SizeType str_num, char *buf, SizeType *offsets, SizeType *lens,
                                      SizeType *res_buckets, hipStream_t stream)
{
    kerFillZero<<<(buckets_num/256)+1,256,0,stream>>>(buckets_num, buf1->getData());
    kerAddData<<<(str_num/256)+1,256,0,stream>>>(str_num, buf, offsets, lens, 0, 
        buckets_num, 
        hashes->getData(), 
        this->lens->getData(), 
        pointers->getData(), 
        res_buckets, 
        buf1->getData());
    CUDA_SAFE_CALL( hipMemcpyFromSymbolAsync ( host_table_is_full_flag->getData(), table_is_full, 
        sizeof(bool), 0, hipMemcpyDeviceToHost, stream ) );
    /// no need to call hipcub::DeviceScan::ExclusiveSum to know buffer size
    /// because we already called it first time in initialization
    hipcub::DeviceScan::ExclusiveSum(buf4->getData(), buf4_sz, buf1->getData(), buf2->getData(), buckets_num, stream);

    kerCopyAddedStrings<<<(str_num/256)+1,256,0,stream>>>( str_num, buf, offsets, lens, res_buckets, 
        str_buf_sz, buf2->getData(), pointers->getData(), str_buf->getData() );

    kerCalcTotalAddedLen<<<1,1,0,stream>>>( buckets_num, buf2->getData(), buf1->getData(), cuda_total_added_num->getData() );
    CUDA_SAFE_CALL( hipMemcpyAsync ( host_total_added_num->getData(), cuda_total_added_num->getData(),
            sizeof(SizeType), hipMemcpyDeviceToHost, stream ) );
    CUDA_SAFE_CALL( hipStreamSynchronize ( stream ) );
    if ((*host_table_is_full_flag)[0]) {
        throw std::runtime_error("CudaStringsHashTable::addData: table is full");
    }
    if (str_buf_sz + (*host_total_added_num)[0] > str_buf_max_sz) throw std::runtime_error("CudaStringsHashTable::addData: table string buffer is full");
    str_buf_sz += (*host_total_added_num)[0];
    //std::cout << "CudaStringsHashTable::addData: calced total new strings length = " << (*host_total_added_num)[0]  << std::endl;
}

void    CudaStringsHashTable::calcOffsets(hipStream_t stream)
{
    kerCalcOffsets<<<(buckets_num/256)+1,256,0,stream>>>(buckets_num, str_buf->getData(), pointers->getData(), offsets->getData() );
}

void    CudaStringsHashTable::mergeToOtherTable(CudaStringsHashTablePtr table, SizeType *res_buckets, hipStream_t stream)
{
    calcOffsets(stream);
    kerFillEmptyResBucket<<<(buckets_num/256)+1,256,0,stream>>>(buckets_num, res_buckets );
    table->addData(buckets_num, str_buf->getData(), offsets->getData(), lens->getData(), res_buckets, stream);
}