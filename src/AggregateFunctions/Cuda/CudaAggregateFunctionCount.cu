#include "hip/hip_runtime.h"
#include <Common/Cuda/CudaAtomics.cuh>
#include <Common/Cuda/CudaSafeCall.h>

#include <AggregateFunctions/Cuda/CudaAggregateFunctionCount.h>

namespace DB
{

__global__ void kerCudaAddBulkCount(
    CudaAggregateFunctionCountData * places,
    ICudaAggregateFunction::CudaSizeType elements_num,
    ICudaAggregateFunction::CudaSizeType * res_buckets)
{
    ICudaAggregateFunction::CudaSizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < elements_num))
        return;
    ICudaAggregateFunction::CudaSizeType res_bucket = res_buckets[i];
    cuda_details::atomicAdd(&(places[res_bucket].count), (UInt64)1);
}


__global__ void kerCudaMergeBulkCount(
    CudaAggregateFunctionCountData * places,
    ICudaAggregateFunction::CudaSizeType elements_num,
    CudaAggregateFunctionCountData * places_from,
    ICudaAggregateFunction::CudaSizeType * res_buckets)
{
    ICudaAggregateFunction::CudaSizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < elements_num))
        return;
    ICudaAggregateFunction::CudaSizeType res_bucket = res_buckets[i];
    if (res_bucket == ~((ICudaAggregateFunction::CudaSizeType)0))
        return;
    cuda_details::atomicAdd(&(places[res_bucket].count), places_from[i].count);
}


void CudaAggregateFunctionCount::cudaAddBulk(
    CudaAggregateDataPtr places,
    CudaColumnStringPtr /*str_column*/,
    CudaSizeType elements_num,
    CudaSizeType * res_buckets,
    char * /*tmp_buf*/,
    hipStream_t stream) const
{
    kerCudaAddBulkCount<<<(elements_num / 256) + 1, 256, 0, stream>>>((CudaAggregateFunctionCountData *)places, elements_num, res_buckets);
}


void CudaAggregateFunctionCount::cudaMergeBulk(
    CudaAggregateDataPtr places,
    CudaSizeType elements_num,
    CudaAggregateDataPtr places_from,
    CudaSizeType * res_buckets,
    hipStream_t stream) const
{
    kerCudaMergeBulkCount<<<(elements_num / 256) + 1, 256, 0, stream>>>(
        (CudaAggregateFunctionCountData *)places, elements_num, (CudaAggregateFunctionCountData *)places_from, res_buckets);
}

}
