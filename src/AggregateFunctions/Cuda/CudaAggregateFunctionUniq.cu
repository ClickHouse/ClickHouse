#include "hip/hip_runtime.h"
#include <Common/Cuda/CudaAtomics.cuh>
#include <Common/Cuda/cudaCalcCityHash64.h>
#include <Common/Cuda/cudaCalcMurmurHash64.h>

#include <AggregateFunctions/Cuda/CudaAggregateFunctionUniq.h>


namespace DB
{

/// the only supported 'type'(T) is String

__global__ void kerCudaInitAggregateData(ICudaAggregateFunction::CudaSizeType places_num, CudaAggregateFunctionUniqHLL12Data * places)
{
    ICudaAggregateFunction::CudaSizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < places_num))
        return;
    (places + i)->initNonzeroData();
}

__global__ void kerCudaAddBulk(
    CudaAggregateFunctionUniqHLL12Data * places,
    ICudaAggregateFunction::CudaSizeType elements_num,
    const UInt64 * hashes,
    ICudaAggregateFunction::CudaSizeType * res_buckets)
{
    ICudaAggregateFunction::CudaSizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < elements_num))
        return;
    UInt64 hash = hashes[i];
    ICudaAggregateFunction::CudaSizeType res_bucket = res_buckets[i];
    places[res_bucket].set.insert(hash);
}

__global__ void kerCudaMergeBulk(
    CudaAggregateFunctionUniqHLL12Data * places,
    ICudaAggregateFunction::CudaSizeType elements_num,
    CudaAggregateFunctionUniqHLL12Data * places_from,
    ICudaAggregateFunction::CudaSizeType * res_buckets)
{
    ICudaAggregateFunction::CudaSizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < elements_num))
        return;
    ICudaAggregateFunction::CudaSizeType res_bucket = res_buckets[i];
    if (res_bucket == ~((ICudaAggregateFunction::CudaSizeType)0))
        return;
    places[res_bucket].set.merge(places_from[i].set);
}

/// The only supported Data here is CudaAggregateFunctionUniqHLL12Data

void CudaAggregateFunctionUniq<String, CudaAggregateFunctionUniqHLL12Data>::cudaInitAggregateData(
    CudaSizeType places_num, CudaAggregateDataPtr places, hipStream_t stream) const
{
    CUDA_SAFE_CALL(hipMemset(places, 0, sizeof(CudaAggregateFunctionUniqHLL12Data) * places_num));
    kerCudaInitAggregateData<<<(places_num / 256) + 1, 256, 0, stream>>>(places_num, (CudaAggregateFunctionUniqHLL12Data *)places);
}

void CudaAggregateFunctionUniq<String, CudaAggregateFunctionUniqHLL12Data>::cudaAddBulk(
    CudaAggregateDataPtr places,
    CudaColumnStringPtr str_column,
    CudaSizeType elements_num,
    CudaSizeType * res_buckets,
    char * tmp_buf,
    hipStream_t stream) const
{
    cudaCalcCityHash64(
        elements_num, str_column->getBuf(), false, str_column->getLens(), str_column->getOffsets(), (UInt64 *)tmp_buf, stream);

    kerCudaAddBulk<<<(elements_num / 256) + 1, 256, 0, stream>>>(
        (CudaAggregateFunctionUniqHLL12Data *)places, elements_num, (UInt64 *)tmp_buf, res_buckets);
}

void CudaAggregateFunctionUniq<String, CudaAggregateFunctionUniqHLL12Data>::cudaMergeBulk(
    CudaAggregateDataPtr places,
    CudaSizeType elements_num,
    CudaAggregateDataPtr places_from,
    CudaSizeType * res_buckets,
    hipStream_t stream) const
{
    kerCudaMergeBulk<<<(elements_num / 256) + 1, 256, 0, stream>>>(
        (CudaAggregateFunctionUniqHLL12Data *)places, elements_num, (CudaAggregateFunctionUniqHLL12Data *)places_from, res_buckets);
}

}
