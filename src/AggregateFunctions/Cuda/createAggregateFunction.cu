
#include <AggregateFunctions/Cuda/CudaAggregateFunctionCount.h>
#include <AggregateFunctions/Cuda/CudaAggregateFunctionUniq.h>
#include <AggregateFunctions/Cuda/ICudaAggregateFunction.h>
#include <AggregateFunctions/Cuda/createAggregateFunction.h>


namespace DB
{

CudaAggregateFunctionPtr createCudaAggregateFunctionCount()
{
    return CudaAggregateFunctionPtr(new CudaAggregateFunctionCount());
}

CudaAggregateFunctionPtr createCudaAggregateFunctionUniq()
{
    return CudaAggregateFunctionPtr(new CudaAggregateFunctionUniq<String, CudaAggregateFunctionUniqHLL12Data>());
}

}
