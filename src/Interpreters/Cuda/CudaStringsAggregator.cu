#include "hip/hip_runtime.h"
#include <chrono>
#include <stdexcept>

#include <Common/Cuda/CudaSafeCall.h>
#include <Common/Cuda/cudaCalcMurmurHash64.h>
#include <Interpreters/Cuda/CudaStringsAggregator.h>

using std::chrono::duration_cast;
using std::chrono::milliseconds;
using std::chrono::steady_clock;

namespace DB
{

CudaStringsAggregator::CudaStringsAggregator(
    int dev_number_,
    size_t chunks_num_,
    UInt32 hash_table_max_size_,
    UInt32 hash_table_str_buffer_max_size_,
    UInt32 buffer_max_str_num_,
    UInt32 buffer_max_size_,
    CudaAggregateFunctionPtr aggregate_function_)
    : dev_number(dev_number_), aggregate_function(aggregate_function_)
{
    CUDA_SAFE_CALL(hipSetDevice(dev_number));
    chunks.resize(chunks_num_);
    /// create cuda streams, allocate host and cuda buffers
    for (size_t i = 0; i < chunks.size(); ++i)
    {
        chunks[i] = WorkChunkInfoPtr(new WorkChunkInfo());
        CUDA_SAFE_CALL(hipStreamCreate(&chunks[i]->stream));
        //chunks[i]->stream = hipStreamPerThread;
        chunks[i]->cuda_hash_table
            = CudaStringsHashTablePtr(new CudaStringsHashTable(hash_table_max_size_, hash_table_str_buffer_max_size_));
        chunks[i]->cuda_buffer_keys = CudaColumnStringPtr(new CudaColumnString(buffer_max_str_num_, buffer_max_size_));
        chunks[i]->cuda_buffer_vals = CudaColumnStringPtr(new CudaColumnString(buffer_max_str_num_, buffer_max_size_));
        chunks[i]->host_buffer_agg_res_keys = CudaHostStringsBufferPtr(new CudaHostStringsBuffer(buffer_max_str_num_, buffer_max_size_));
        chunks[i]->group_nums.resize(buffer_max_str_num_);
        chunks[i]->group_agg_res = CudaArrayPtr<char>(new CudaArray<char>(hash_table_max_size_ * aggregate_function->cudaSizeOfData()));
        chunks[i]->host_group_agg_res
            = CudaHostPinnedArrayPtr<char>(new CudaHostPinnedArray<char>(hash_table_max_size_ * aggregate_function->cudaSizeOfData()));
        chunks[i]->agg_tmp_buf
            = CudaArrayPtr<char>(new CudaArray<char>(aggregate_function->cudaSizeOfAddBulkInternalBuf(buffer_max_str_num_)));
    }
    CUDA_SAFE_CALL(hipStreamCreate(&copy_stream));
    std::cout << "CudaStringsAggregator created" << std::endl;
}

struct ProcessChunkParams
{
    ProcessChunkParams(CudaStringsAggregator * agg_, size_t i_) : agg(agg_), i(i_) { }

    CudaStringsAggregator * agg;
    size_t i;
};

void callProcessChunk(ProcessChunkParams params)
{
    params.agg->processChunk(params.i);
}

void CudaStringsAggregator::startProcessing()
{
    /// start processing threads
    for (size_t i = 0; i < chunks.size(); ++i)
    {
        chunks[i]->cuda_processing_state = false;
        chunks[i]->t = std::thread(callProcessChunk, ProcessChunkParams(this, i));
    }
    /// set current buffer for data appending
    curr_filling_chunk = 0;
    is_vals_needed = aggregate_function->isDataNeeded();
}


void CudaStringsAggregator::queueData(
    size_t str_num,
    size_t str_buf_sz,
    const char * str_buf,
    const OffsetType * offsets,
    size_t vals_str_buf_sz,
    const char * vals_str_buf,
    const OffsetType * vals_offsets)
{
    while (1)
    {
        if (tryQueueData(str_num, str_buf_sz, str_buf, offsets, vals_str_buf_sz, vals_str_buf, vals_offsets))
            return;
    }
}

void CudaStringsAggregator::waitQueueData() const
{
    CUDA_SAFE_CALL(hipStreamSynchronize(copy_stream));
}

void CudaStringsAggregator::waitProcessed()
{
    {
        std::unique_lock<std::mutex> lck(chunks[curr_filling_chunk]->cuda_buffer_mtx);
        chunks[curr_filling_chunk]->cv_cuda_processing_end.wait(lck, [this] { return !chunks[curr_filling_chunk]->cuda_processing_state; });

        if (!chunks[curr_filling_chunk]->cuda_buffer_keys->empty())
        {
            chunks[curr_filling_chunk]->cuda_processing_state = true;
            chunks[curr_filling_chunk]->cv_buffer_append_end.notify_one();
        }
    }

    /// wait till host to gpu copy ends, 'send' empty buffer to signal end of data
    for (size_t i = 0; i < chunks.size(); ++i)
    {
        std::unique_lock<std::mutex> lck(chunks[i]->cuda_buffer_mtx);
        chunks[i]->cv_cuda_processing_end.wait(lck, [this, i] { return !chunks[i]->cuda_processing_state; });
        if (!chunks[i]->cuda_buffer_keys->empty())
            throw std::logic_error("CudaStringsAggregator: host buffer is not empty after transfer");
        //setting cuda_processing_state with empty buffer means end of processing
        chunks[i]->cuda_processing_state = true;
        chunks[i]->cv_buffer_append_end.notify_one();
    }

    /// wait for processes for termination
    for (size_t i = 0; i < chunks.size(); ++i)
    {
        chunks[i]->t.join();
    }

    /// combine data from different chunks
    for (size_t i = 1; i < chunks.size(); ++i)
    {
        chunks[i]->cuda_hash_table->mergeToOtherTable(
            chunks[0]->cuda_hash_table, thrust::raw_pointer_cast(chunks[i]->group_nums.data()), chunks[0]->stream);

        aggregate_function->cudaMergeBulk(
            chunks[0]->group_agg_res->getData(),
            chunks[i]->cuda_hash_table->getBucketsNum(),
            chunks[i]->group_agg_res->getData(),
            thrust::raw_pointer_cast(chunks[i]->group_nums.data()),
            chunks[0]->stream);
    }

    CUDA_SAFE_CALL(hipMemcpyAsync(
        chunks[0]->host_buffer_agg_res_keys->getBuf(),
        chunks[0]->cuda_hash_table->getStrBuf(),
        chunks[0]->cuda_hash_table->getStrBufSz(),
        hipMemcpyDeviceToHost,
        chunks[0]->stream));
    /// TODO get rid of sizeof(UInt32) in all following hipMemcpyAsync!!
    CUDA_SAFE_CALL(hipMemcpyAsync(
        chunks[0]->host_buffer_agg_res_keys->getLens(),
        chunks[0]->cuda_hash_table->getLens(),
        chunks[0]->cuda_hash_table->getBucketsNum() * sizeof(UInt32),
        hipMemcpyDeviceToHost,
        chunks[0]->stream));
    CUDA_SAFE_CALL(hipMemcpyAsync(
        chunks[0]->host_buffer_agg_res_keys->getOffsets(),
        chunks[0]->cuda_hash_table->getOffsets(),
        chunks[0]->cuda_hash_table->getBucketsNum() * sizeof(UInt32),
        hipMemcpyDeviceToHost,
        chunks[0]->stream));
    CUDA_SAFE_CALL(hipMemcpyAsync(
        chunks[0]->host_group_agg_res->getData(),
        chunks[0]->group_agg_res->getData(),
        chunks[0]->cuda_hash_table->getBucketsNum() * aggregate_function->cudaSizeOfData(),
        hipMemcpyDeviceToHost,
        chunks[0]->stream));

    CUDA_SAFE_CALL(hipStreamSynchronize(chunks[0]->stream));

    auto host_e1 = steady_clock::now();
    for (size_t j = 0; j < chunks[0]->cuda_hash_table->getBucketsNum(); ++j)
    {
        UInt32 len = chunks[0]->host_buffer_agg_res_keys->getLens()[j], offset = chunks[0]->host_buffer_agg_res_keys->getOffsets()[j];
        if (len == DBMS_CUDA_EMPTY_LEN_VAL)
            continue;
        std::string key_str(chunks[0]->host_buffer_agg_res_keys->getBuf() + offset, len - 1);
        CudaAggregateDataPtr res = chunks[0]->host_group_agg_res->getData() + j * aggregate_function->cudaSizeOfData();

        auto it = chunks[0]->agg_result.find(key_str);
        if (it == chunks[0]->agg_result.end())
        {
            chunks[0]->agg_result[key_str] = res;
        }
        else
        {
            throw std::logic_error("CudaStringsAggregator::waitProcessed: seems there are duplicates in GPU table");
        }
    }

    auto host_e2 = steady_clock::now();
    auto host_t = duration_cast<milliseconds>(host_e2 - host_e1);
    std::cout << "CudaStringsAggregator::waitProcessed: time for placing data into cpu hash table " << host_t.count() << "ms" << std::endl;
}


bool CudaStringsAggregator::tryQueueData(
    size_t str_num,
    size_t str_buf_sz,
    const char * str_buf,
    const OffsetType * offsets,
    size_t vals_str_buf_sz,
    const char * vals_str_buf,
    const OffsetType * vals_offsets)
{
    std::unique_lock<std::mutex> lck(chunks[curr_filling_chunk]->cuda_buffer_mtx);
    chunks[curr_filling_chunk]->cv_cuda_processing_end.wait(lck, [this] { return !chunks[curr_filling_chunk]->cuda_processing_state; });

    if (chunks[curr_filling_chunk]->cuda_buffer_keys->hasSpace(str_num, str_buf_sz)
        && chunks[curr_filling_chunk]->cuda_buffer_vals->hasSpace(str_num, vals_str_buf_sz))
    {
        chunks[curr_filling_chunk]->cuda_buffer_keys->addData(str_num, str_buf_sz, str_buf, offsets, copy_stream);
        if (is_vals_needed)
            chunks[curr_filling_chunk]->cuda_buffer_vals->addData(str_num, vals_str_buf_sz, vals_str_buf, vals_offsets, copy_stream);
        return true;
    }
    else
    {
        if (chunks[curr_filling_chunk]->cuda_buffer_keys->empty())
            throw std::runtime_error("CudaStringsAggregator: seems there is not enough space in buffer");
        waitQueueData();
        chunks[curr_filling_chunk]->cuda_processing_state = true;
        chunks[curr_filling_chunk]->cv_buffer_append_end.notify_one();
        curr_filling_chunk = (curr_filling_chunk + 1) % chunks.size();
        return false;
    }
}


__global__ void kerFillMaxHash(UInt32 str_num, UInt32 max_str_num, UInt64 * hashes)
{
    UInt32 i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < str_num)
        return;
    if (!(i < max_str_num))
        return;

    hashes[i] = 0xFFFFFFFFFFFFFFFF;
}


void CudaStringsAggregator::processChunk(size_t i)
{
    CUDA_SAFE_CALL(hipSetDevice(dev_number));

    chunks[i]->cuda_hash_table->erase(chunks[i]->stream);
    aggregate_function->cudaInitAggregateData(
        chunks[i]->cuda_hash_table->getBucketsNum(), chunks[i]->group_agg_res->getData(), chunks[i]->stream);

    while (1)
    {
        {
            std::cout << "CudaStringsAggregator::processChunk(i = " << i << "): waiting data..." << std::endl;
            std::unique_lock<std::mutex> lck(chunks[i]->cuda_buffer_mtx);
            chunks[i]->cv_buffer_append_end.wait(lck, [this, i] { return chunks[i]->cuda_processing_state; });
            /// we agreed that empty buffer means end of processing
            if (chunks[i]->cuda_buffer_keys->empty())
                break;

            std::cout << "CudaStringsAggregator::processChunk(i = " << i << "): calc Lengths" << std::endl;
            chunks[i]->cuda_buffer_keys->calcLengths(chunks[i]->stream);
            if (is_vals_needed)
                chunks[i]->cuda_buffer_vals->calcLengths(chunks[i]->stream);

            size_t str_num = chunks[i]->cuda_buffer_keys->getStrNum();
            chunks[i]->cuda_hash_table->addData(
                str_num,
                chunks[i]->cuda_buffer_keys->getBuf(),
                chunks[i]->cuda_buffer_keys->getOffsets(),
                chunks[i]->cuda_buffer_keys->getLens(),
                thrust::raw_pointer_cast(chunks[i]->group_nums.data()),
                chunks[i]->stream);
            aggregate_function->cudaAddBulk(
                chunks[i]->group_agg_res->getData(),
                chunks[i]->cuda_buffer_vals,
                str_num,
                thrust::raw_pointer_cast(chunks[i]->group_nums.data()),
                chunks[i]->agg_tmp_buf->getData(),
                chunks[i]->stream);

            chunks[i]->cuda_buffer_keys->reset();
            chunks[i]->cuda_buffer_vals->reset();
            chunks[i]->cuda_processing_state = false;
            chunks[i]->cv_cuda_processing_end.notify_one();
        }
    }

    chunks[i]->cuda_hash_table->calcOffsets(chunks[i]->stream);
    CUDA_SAFE_CALL(hipStreamSynchronize(chunks[i]->stream));
}


CudaStringsAggregator::~CudaStringsAggregator()
{
}

}
