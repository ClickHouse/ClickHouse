#include "hip/hip_runtime.h"
#include <cassert>
#include <stdexcept>

#include <Columns/Cuda/CudaColumnString.h>
#include <Common/Cuda/CudaSafeCall.h>


namespace DB
{

CudaColumnString::CudaColumnString(size_t max_str_num_, size_t max_sz_) : max_str_num(max_str_num_), max_sz(max_sz_), str_num(0), sz(0)
{
    if ((max_str_num_ == 0) || (max_sz_ == 0))
        throw std::logic_error("CudaColumnString: try to create zero size buffer");
    hipError_t err;
    err = hipMalloc((void **)&buf, max_sz * sizeof(char));
    if (err != hipSuccess)
        throw std::runtime_error("CudaColumnString: failed to alloc cuda memory for strings");
    err = hipMalloc((void **)&lens, max_str_num * sizeof(UInt32));
    if (err != hipSuccess)
    {
        CUDA_SAFE_CALL_NOTHROW(hipFree(buf));
        throw std::runtime_error("CudaColumnString: failed to alloc cuda memory for lengths buffer");
    }
    err = hipMalloc((void **)&offsets, max_str_num * sizeof(UInt32));
    if (err != hipSuccess)
    {
        CUDA_SAFE_CALL_NOTHROW(hipFree(buf));
        CUDA_SAFE_CALL_NOTHROW(hipFree(lens));
        throw std::runtime_error("CudaColumnString: failed to alloc cuda memory for offsets buffer");
    }

    err = hipMalloc((void **)&offsets64, max_str_num * sizeof(UInt64));
    if (err != hipSuccess)
    {
        CUDA_SAFE_CALL_NOTHROW(hipFree(buf));
        CUDA_SAFE_CALL_NOTHROW(hipFree(lens));
        CUDA_SAFE_CALL_NOTHROW(hipFree(offsets));
        throw std::runtime_error("CudaColumnString: failed to alloc cuda memory for offsets64 buffer");
    }
}

bool CudaColumnString::hasSpace(size_t str_num_, size_t str_buf_sz_) const
{
    if (str_num + str_num_ > max_str_num)
        return false;
    if (sz + str_buf_sz_ > max_sz)
        return false;
    return true;
}

void CudaColumnString::addData(size_t str_num_, size_t str_buf_sz_, const char * str_buf_, const UInt64 * offsets_, hipStream_t stream)
{
    CUDA_SAFE_CALL(hipMemcpyAsync(getBuf() + sz, str_buf_, str_buf_sz_, hipMemcpyHostToDevice, stream));
    // TODO do something with sizeof(UInt64)
    CUDA_SAFE_CALL(hipMemcpyAsync(getOffsets64() + str_num, offsets_, str_num_ * sizeof(UInt64), hipMemcpyHostToDevice, stream));

    str_num += str_num_;
    sz += str_buf_sz_;
    blocks_sizes.push_back(str_num_);
    blocks_buf_sizes.push_back(str_buf_sz_);
}

void CudaColumnString::setSize(size_t str_num_, size_t sz_)
{
    str_num = str_num_;
    sz = sz_;
}

__global__ void kerCalcLengths(
    UInt32 block_begin, UInt32 block_size, UInt32 block_offset, UInt32 block_buf_size, UInt64 * offsets64, UInt32 * lens, UInt32 * offsets)
{
    UInt32 i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < block_size))
        return;
    UInt32 local_offset, local_offset_next = offsets64[block_begin + i];
    if (i > 0)
        local_offset = offsets64[block_begin + i - 1];
    else
        local_offset = 0;

    offsets[block_begin + i] = local_offset + block_offset;
    lens[block_begin + i] = local_offset_next - local_offset;
}

void CudaColumnString::reset()
{
    str_num = 0;
    sz = 0;
    blocks_sizes.clear();
    blocks_buf_sizes.clear();
}

void CudaColumnString::calcLengths(hipStream_t stream)
{
    assert(blocks_sizes.size() == blocks_buf_sizes.size());
    UInt32 block_begin = 0, block_offset = 0;
    for (size_t i = 0; i < blocks_sizes.size(); ++i)
    {
        kerCalcLengths<<<(blocks_sizes[i] / 256) + 1, 256, 0, stream>>>(
            block_begin, blocks_sizes[i], block_offset, blocks_buf_sizes[i], offsets64, lens, offsets);
        block_begin += blocks_sizes[i];
        block_offset += blocks_buf_sizes[i];
    }
}

CudaColumnString::~CudaColumnString()
{
    CUDA_SAFE_CALL_NOTHROW(hipFree(buf));
    CUDA_SAFE_CALL_NOTHROW(hipFree(lens));
    CUDA_SAFE_CALL_NOTHROW(hipFree(offsets));
    CUDA_SAFE_CALL_NOTHROW(hipFree(offsets64));
}

}
