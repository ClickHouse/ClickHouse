
#define USE_PARALLEL_MEMCPY

#include <chrono>
#include <iostream>
#include "CudaHostStringsBuffer.h"
#include "parallelMemcpy.h"

namespace DB
{

using std::chrono::duration_cast;
using std::chrono::milliseconds;
using std::chrono::steady_clock;

CudaHostStringsBuffer::CudaHostStringsBuffer(size_t max_str_num_, size_t max_sz_, bool has_lens_, bool has_offsets_, bool has_offsets64_)
    : max_str_num(max_str_num_)
    , max_sz(max_sz_)
    , str_num(0)
    , sz(0)
    , has_lens(has_lens_)
    , has_offsets(has_offsets_)
    , has_offsets64(has_offsets64_)
{
    buf = CudaHostPinnedArrayPtr<char>(new CudaHostPinnedArray<char>(max_sz));
    if (has_lens)
        lens = CudaHostPinnedArrayPtr<UInt32>(new CudaHostPinnedArray<UInt32>(max_str_num));
    if (has_offsets)
        offsets = CudaHostPinnedArrayPtr<UInt32>(new CudaHostPinnedArray<UInt32>(max_str_num));
    if (has_offsets64)
        offsets64 = CudaHostPinnedArrayPtr<UInt64>(new CudaHostPinnedArray<UInt64>(max_str_num));
}

bool CudaHostStringsBuffer::hasSpace(size_t str_num_, size_t str_buf_sz_) const
{
    if (str_num + str_num_ > max_str_num)
        return false;
    if (sz + str_buf_sz_ > max_sz)
        return false;
    return true;
}

void CudaHostStringsBuffer::addData(
    size_t str_num_, size_t str_buf_sz_, const char * str_buf_, const UInt64 * offsets_, size_t memcpy_threads_num_)
{
    auto host_e1 = steady_clock::now();
#ifdef USE_PARALLEL_MEMCPY
    parallelMemcpy((char *)(buf->getData() + sz), (const char *)str_buf_, str_buf_sz_, memcpy_threads_num_);
    parallelMemcpy((char *)(offsets64->getData() + str_num), (const char *)offsets_, str_num_ * sizeof(UInt64), memcpy_threads_num_);
#else
    memcpy(buf->getData() + sz, str_buf_, str_buf_sz_);
    memcpy(offsets64->getData() + str_num, offsets_, str_num_ * sizeof(UInt64));
#endif
    auto host_e2 = steady_clock::now();
    auto host_t = duration_cast<milliseconds>(host_e2 - host_e1);
    str_num += str_num_;
    sz += str_buf_sz_;
    blocks_sizes.push_back(str_num_);
    blocks_buf_sizes.push_back(str_buf_sz_);
}

void CudaHostStringsBuffer::setSize(size_t str_num_, size_t sz_)
{
    str_num = str_num_;
    sz = sz_;
}

void CudaHostStringsBuffer::reset()
{
    str_num = 0;
    sz = 0;
    blocks_sizes.clear();
    blocks_buf_sizes.clear();
}

}
